#include "hip/hip_runtime.h"
#include <optix_world.h>
#include <optix_device.h>

#include "colvillea/Device/Toolkit/NvRandom.h"
#include "colvillea/Device/Toolkit/Utility.h"
#include "colvillea/Device/Toolkit/CommonStructs.h"
#include "colvillea/Device/Sampler/Sampler.h"
#include "colvillea/Device/Filter/Filter.h"


using namespace optix;

//////////////////////////////////////////////////////////////////////////
//Forward declarations:
//system variables related:->Context
rtBuffer<float4, 2>	        sysOutputBuffer;         /*final result buffer. Necessary to keep Alpha=1*/
rtBuffer<float4, 2>         sysHDRBuffer;            /* Final result buffer for exporting to OpenEXR. */
rtBuffer<float, 2>          sysSampleWeightedSum;
rtBuffer<float4, 2>         sysCurrResultBuffer;     /*the sum of weighted radiance(f(dx,dy)*Li) with 
												       respect to the current iteration;
													   A is useless, note that for RenderView display,
													   it's necessary to keep as 1.f to prevent from 
													   alpha cutout by Dear Imgui*/
rtBuffer<float, 2>          sysCurrWeightedSumBuffer;/*weightedSum buffer with respect to the current
												       iteration*/

//rtDeclareVariable(float,    sysFilterGaussianAlpha, ,) = 0.25f;//Gaussian filter alpha paramter
//rtDeclareVariable(float,    sysFilterWidth, ,) = 1.f;//Gaussian filter width>=1.f

rtDeclareVariable(float,    sysSceneEpsilon, , );
rtDeclareVariable(rtObject, sysTopObject, , );


#ifndef TWRT_DECLARE_SYSLAUNCH
#define TWRT_DECLARE_SYSLAUNCH
rtDeclareVariable(uint2,    sysLaunch_Dim,       rtLaunchDim, );
rtDeclareVariable(uint2,    sysLaunch_index,     rtLaunchIndex, );
#endif

//camera variables related:->Program
rtDeclareVariable(Matrix4x4, RasterToCamera, , );
rtDeclareVariable(Matrix4x4, CameraToWorld, , );
rtDeclareVariable(float,     focalDistance, ,)=0.0f;
rtDeclareVariable(float,     lensRadius, ,)=0.0f;



namespace TwUtil
{
	/*
	 * @brief degamma function converts linear color to sRGB color
	 * for display
	 * @param src input value, alpha channel is not affected
	 * @return corresponding sRGB encoded color in float4. Alpha channel
	 * is left unchanged.
	 * @ref https://www.khronos.org/registry/OpenGL/extensions/EXT/EXT_texture_sRGB_decode.txt
	 * @see convertsRGBToLinear
	 **/
	static __device__ __inline__ float4 convertFromLinearTosRGB(const float4 &src)
	{
		float4 dst = src;
		dst.x = (dst.x < 0.0031308f) ? dst.x*12.92f : (1.055f * powf(dst.x, 0.41666f) - 0.055f);
		dst.y = (dst.y < 0.0031308f) ? dst.y*12.92f : (1.055f * powf(dst.y, 0.41666f) - 0.055f);
		dst.z = (dst.z < 0.0031308f) ? dst.z*12.92f : (1.055f * powf(dst.z, 0.41666f) - 0.055f);
		return dst;
	}

	/*
	 * @brief converts one of the sRGB color channel to linear
	 * @param src input value
	 * @return corresponding linear space color channel
	 * @ref https://www.khronos.org/registry/OpenGL/extensions/EXT/EXT_texture_sRGB_decode.txt
	 * @see convertFromLinearTosRGB
	 **/
	static __device__ __inline__ float convertsRGBToLinear(const float &src)
	{
		if (src <= 0.f)
			return 0;
		if (src >= 1.f)
			return 1.f;
		if (src <= 0.04045f)
			return src / 12.92f;
		return pow((src + 0.055f) / 1.055f, 2.4f);
	};

	/*
	 * @brief converts sRGB color to linear color
	 * @param src input value, alpha channel is not affected
	 * @return corresponding linear space color in float4. Alpha channel
	 * is left unchanged.
	 * @ref https://www.khronos.org/registry/OpenGL/extensions/EXT/EXT_texture_sRGB_decode.txt
	 * @see convertFromLinearTosRGB
	 **/
	static __device__ __inline__ float4 convertsRGBToLinear(const float4 &src)
	{
		return make_float4(convertsRGBToLinear(src.x), convertsRGBToLinear(src.y), convertsRGBToLinear(src.z), 1.f);
	}
};

//////////////////////////////////////////////////////////////////////////
//Program definitions:
RT_PROGRAM void RayGeneration_PinholeCamera()
{
    /* Make sampler and preprocess. */
    GPUSampler localSampler;  /* GPUSampler is a union type, use out parameter instead of return value to avoid copying construct on union, which could lead to problems. */
    makeSampler(RayTracingPipelinePhase::RayGeneration, localSampler);

    /* Fetch camera samples lying on [0,1]^2. */
    float2 qmcSamples = Get2D(&localSampler);

	/* Calculate filmSamples in continuous coordinates. */
	float2 pFilm = qmcSamples + make_float2(static_cast<float>(sysLaunch_index.x), static_cast<float>(sysLaunch_index.y));

	/* Generate ray from camera. */
	float3 rayOrg = make_float3(0.f);
	float3 rayDir = rayOrg;
    if (lensRadius > 0.f)
    {
        float2 lensSamples = Get2D(&localSampler);
        TwUtil::GenerateRay(pFilm, rayOrg, rayDir, RasterToCamera, CameraToWorld, lensRadius, focalDistance, &lensSamples);
    }
    else
    {
        TwUtil::GenerateRay(pFilm, rayOrg, rayDir, RasterToCamera, CameraToWorld, lensRadius, focalDistance, nullptr);
    }

	/* Make ray and trace, goint to next raytracing pipeline phase. */
	Ray ray = make_Ray(rayOrg, rayDir, toUnderlyingValue(RayType::Radiance), sysSceneEpsilon, RT_DEFAULT_MAX);

    CommonStructs::PerRayData_radiance prdRadiance;
	prdRadiance.radiance = make_float4(0.f);

	rtTrace<CommonStructs::PerRayData_radiance>(sysTopObject, ray, prdRadiance, RT_VISIBILITY_ALL, RT_RAY_FLAG_DISABLE_ANYHIT);

	/*--------------------------------------------------------------------------------*/
	/*----Perform filtering and reconstruction so as to write to the output buffer----*/
	/*--------------------------------------------------------------------------------*/

    /* If filter width <= 0.5f, one sample could only contribute to one pixel
       -- and there is no chance that two samples not in the same pixel will
       -- contribute to the same pixel. So atomic operation could be saved for
       -- efficenciy. */
    float filterWidth = GetFilterWidth();
    if(filterWidth <= 0.5f)
    {
        float currentWeight = EvaluateFilter(qmcSamples.x - 0.5f, qmcSamples.y - 0.5f);

        float4 &currLi = prdRadiance.radiance;
        /*ignore alpha channel*/
        sysCurrResultBuffer[sysLaunch_index].x    += currLi.x * currentWeight;
        sysCurrResultBuffer[sysLaunch_index].y    += currLi.y * currentWeight;
        sysCurrResultBuffer[sysLaunch_index].z    += currLi.z * currentWeight;
        sysCurrWeightedSumBuffer[sysLaunch_index] += currentWeight;
    }
    else
    {
        /* Compute pFilm's raster extent
         * --1.get film sample's discrete coordinates. */
        float2 dCoordsSample = pFilm - 0.5f;

        /*--2.search around the filterWidth for raster pixel boundary*/
        int2 pMin = TwUtil::ceilf2i(dCoordsSample - filterWidth);
        int2 pMax = TwUtil::floorf2i(dCoordsSample + filterWidth);

        /*--3.check for film extent*/
        pMin.x = max(pMin.x, 0);                       pMin.y = max(pMin.y, 0);
        pMax.x = min(pMax.x, sysLaunch_Dim.x - 1);     pMax.y = min(pMax.y, sysLaunch_Dim.y - 1);

        if ((pMax.x - pMin.x) < 0 || (pMax.y - pMin.y) < 0)
        {
            rtPrintf("invalid samples:%f %f\n", dCoordsSample.x, dCoordsSample.y);
        }

        /* Loop over raster pixel and add sample with filtering operation. */
        for (int y = pMin.y; y <= pMax.y; ++y)
        {
            for (int x = pMin.x; x <= pMax.x; ++x)
            {
                /*not necessary to distinguish first iteration, because one sample
                 *could possibly contribute to multiple pixels so the 0th iteration doesn't
                 *have a specialized meaning. Instead, we use the modified version of progressive
                 *weighted average formula.*/

                 /*Pass 1:accumulate sysCurrResultBuffer with f(dx,dy)*Li and sysCurrWeightedSumBuffer with f(dx,dy)*/
                uint2 pixelIndex = make_uint2(x, y);
                float currentWeight = EvaluateFilter(x - dCoordsSample.x, y - dCoordsSample.y);

                float4 &currLi = prdRadiance.radiance;
                /*ignore alpha channel*/
                atomicAdd(&sysCurrResultBuffer[pixelIndex].x, currLi.x * currentWeight);
                atomicAdd(&sysCurrResultBuffer[pixelIndex].y, currLi.y * currentWeight);
                atomicAdd(&sysCurrResultBuffer[pixelIndex].z, currLi.z * currentWeight);
                atomicAdd(&sysCurrWeightedSumBuffer[pixelIndex], currentWeight);
            }
        }
    }
}

//////////////////////////////////////////////////////////////////////////
//Initialize outputbuffer and sampleWeightedSum buffer, much more efficient than using serilized "for" on host 
RT_PROGRAM void RayGeneration_InitializeFilter()
{
	sysCurrResultBuffer[sysLaunch_index] = sysOutputBuffer[sysLaunch_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	sysCurrWeightedSumBuffer[sysLaunch_index] = sysSampleWeightedSum[sysLaunch_index] = 0.0f;
}

//////////////////////////////////////////////////////////////////////////
//Perform filtering
RT_PROGRAM void RayGeneration_Filter()
{
	/* Perform gamma correction to resolve correct linear color for computation. */
	sysOutputBuffer[sysLaunch_index] = convertsRGBToLinear(sysOutputBuffer[sysLaunch_index]);
	sysOutputBuffer[sysLaunch_index] = (sysOutputBuffer[sysLaunch_index] * sysSampleWeightedSum[sysLaunch_index] + sysCurrResultBuffer[sysLaunch_index]) / 
                                       (sysSampleWeightedSum[sysLaunch_index] + sysCurrWeightedSumBuffer[sysLaunch_index]);
    sysHDRBuffer[sysLaunch_index]    = (sysHDRBuffer[sysLaunch_index] * sysSampleWeightedSum[sysLaunch_index] + sysCurrResultBuffer[sysLaunch_index]) / 
                                       (sysSampleWeightedSum[sysLaunch_index] + sysCurrWeightedSumBuffer[sysLaunch_index]);

	sysSampleWeightedSum[sysLaunch_index] += sysCurrWeightedSumBuffer[sysLaunch_index];

    /* Ensure w component of output buffer is 1.0f in case of being transparent
     * -- in RenderView. */
    sysOutputBuffer[sysLaunch_index].w = 1.f;
    sysHDRBuffer[sysLaunch_index].w    = 1.f;

    /* Prevent from precision error. Note that some filters (such as Gaussian)
     * -- could have zero sample weight to a pixel, which may cause that for some
     * -- pixels, they couldn't have any sample weight contribution at all at first
     * -- launch (almost only one sample distributes sparsely per pixel in the situation).
     * But after a few iterations, all pixels are able to have sample weight contribution.
     * For this reason, we skip the NaN values in output buffer to give a chance for
     * later accumulation. */
    if (isnan(sysOutputBuffer[sysLaunch_index].x) || isnan(sysOutputBuffer[sysLaunch_index].y) || isnan(sysOutputBuffer[sysLaunch_index].z) || 
        isnan(sysHDRBuffer[sysLaunch_index].x)    || isnan(sysHDRBuffer[sysLaunch_index].y)    || isnan(sysHDRBuffer[sysLaunch_index].z))
    {
        sysOutputBuffer[sysLaunch_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
        sysHDRBuffer[sysLaunch_index]    = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
    }

    #if 0
    if (isnan(sysOutputBuffer[sysLaunch_index].x) || isnan(sysOutputBuffer[sysLaunch_index].y) || isnan(sysOutputBuffer[sysLaunch_index].z) || (sysOutputBuffer[sysLaunch_index].x < 0.f || sysOutputBuffer[sysLaunch_index].y < 0.f || sysOutputBuffer[sysLaunch_index].z < 0.f))
    {
        rtPrintf("%d %d out=%f * %f + %f / (%f + %f)\n", sysLaunch_index.x,sysLaunch_index.y,
            sysOutputBuffer[sysLaunch_index].x,
            sysSampleWeightedSum[sysLaunch_index], 
            sysCurrResultBuffer[sysLaunch_index].x, 
            sysSampleWeightedSum[sysLaunch_index], 
            sysCurrWeightedSumBuffer[sysLaunch_index]);
    }
#endif // 
    

	/*clear current buffer for next iteration*/
	sysCurrResultBuffer[sysLaunch_index] = make_float4(0.0f, 0.0f, 0.0f, 1.0f);
	sysCurrWeightedSumBuffer[sysLaunch_index] = 0.0f;

	/* Perform "degamma" operation converting linear color to sRGB for diplaying in RenderView. */
	sysOutputBuffer[sysLaunch_index] = convertFromLinearTosRGB(sysOutputBuffer[sysLaunch_index]);
}


RT_PROGRAM void Exception_Default()
{
	const unsigned int code = rtGetExceptionCode();
	rtPrintf("[Exception]Caught exception 0x%X at launch index (%d,%d)\n", code, sysLaunch_index.x, sysLaunch_index.y);
	rtPrintExceptionDetails();
	sysOutputBuffer[sysLaunch_index] = make_float4(1000.0f, 0.0f, 0.0f, 1.0f);
}


